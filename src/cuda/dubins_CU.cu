#include "hip/hip_runtime.h"
#include <dubins_CU.hh>

#define pow2(x) x*x
#define CUDA_Epsi 1e-10
#define CUDA_DInf 0x7ff0000000000000 
#define CUDA_FInf 0x7f800000 

#include<limits>
#include<iostream>
using namespace std;

#define DInf numeric_limits<double>::infinity()

__device__ bool CUDA_equal (double x, double y, double epsi=CUDA_Epsi){ return fabs(x-y)<epsi;}

__device__ __host__ double mod2pi (double angle){
	while(angle>=2*M_PI){
		angle-=(M_PI*2);
	}
	while(angle<0){
		angle+=(M_PI*2);
	}
	return angle;
}

__global__ void LSL (double th0, double th1, double _kmax, double* ret)
{
	double C=cos(th1)-cos(th0);
	double S=2*_kmax+sin(th0)-sin(th1);
	double tan2=atan2(C, S);

	double temp1=2+4*pow2(_kmax)-2*cos(th0-th1)+4*_kmax*(sin(th0)-sin(th1));

	if (temp1<0){
	  ret[0]=-1;
	  return;
	}

	double invK=1/_kmax;
	double sc_s1=mod2pi(tan2-th0)*invK;
	double sc_s2=invK*sqrt(temp1);
	double sc_s3=mod2pi(th1-tan2)*invK;

	ret[0]=sc_s1;
	ret[1]=sc_s2;
	ret[2]=sc_s3;
	// printf("in_LSL_dev th0: %f, th1: %f kmax: %f C: %f S: %f temp1: %f invk: %f ret: %f %f %f\n", th0, th1, _kmax, C, S, temp1, invK, ret[0], ret[1], ret[2]);
}

__global__ void RSR (double th0, double th1, double _kmax, double* ret)
{
	double C=cos(th0)-cos(th1);
	double S=2*_kmax-sin(th0)+sin(th1);

	double temp1=2+4*pow2(_kmax)-2*cos(th0-th1)-4*_kmax*(sin(th0)-sin(th1));

	if (temp1<0){
	  ret[0]=-1;
	  return;
	}

	double invK=1/_kmax;
	double sc_s1=mod2pi(th0-atan2(C,S))*invK;
	double sc_s2=invK*sqrt(temp1);
	double sc_s3=mod2pi(atan2(C,S)-th1)*invK;

	ret[0]=sc_s1;
	ret[1]=sc_s2;
	ret[2]=sc_s3;
	// printf("in_RSR_dev th0: %f, th1: %f kmax: %f C: %f S: %f temp1: %f invk: %f ret: %f %f %f\n", th0, th1, _kmax, C, S, temp1, invK, ret[0], ret[1], ret[2]);
}

__global__ void LSR (double th0, double th1, double _kmax, double* ret)
{    
	double C = cos(th0)+cos(th1);
	double S=2*_kmax+sin(th0)+sin(th1);

	double temp1=-2+4*pow2(_kmax)+2*cos(th0-th1)+4*_kmax*(sin(th0)+sin(th1));

	if (temp1<0){
	  ret[0]=-1;
	  return;
	}

	double invK=1/_kmax;

	double sc_s2=invK*sqrt(temp1);
	double sc_s1= mod2pi(atan2(-C,S)-atan2(-2.0, _kmax*sc_s2)-th0)*invK;
	double sc_s3= mod2pi(atan2(-C,S)-atan2(-2.0, _kmax*sc_s2)-th1)*invK;

	ret[0]=sc_s1;
	ret[1]=sc_s2;
	ret[2]=sc_s3;
	// printf("in_LSR_dev th0: %f, th1: %f kmax: %f C: %f S: %f temp1: %f invk: %f ret: %f %f %f\n", th0, th1, _kmax, C, S, temp1, invK, ret[0], ret[1], ret[2]);
}

__global__ void RSL (double th0, double th1, double _kmax, double* ret)
{
	double C = cos(th0)+cos(th1);
	double S=2*_kmax-sin(th0)-sin(th1);

	double temp1=-2+4*pow2(_kmax)+2*cos(th0-th1)-4*_kmax*(sin(th0)+sin(th1));
	if (temp1<0){
	  ret[0]=-1;
	  return;
	}

	double invK=1/_kmax;

	double sc_s2=invK*sqrt(temp1);
	double sc_s1= mod2pi(th0-atan2(C,S)+atan2(2.0, _kmax*sc_s2))*invK;
	double sc_s3= mod2pi(th1-atan2(C,S)+atan2(2.0, _kmax*sc_s2))*invK;

	ret[0]=sc_s1;
	ret[1]=sc_s2;
	ret[2]=sc_s3;
	// printf("in_RSL_dev th0: %f, th1: %f kmax: %f C: %f S: %f temp1: %f invk: %f ret: %f %f %f\n", th0, th1, _kmax, C, S, temp1, invK, ret[0], ret[1], ret[2]);
}

__global__ void RLR (double th0, double th1, double _kmax, double* ret)
{
	double C=cos(th0)-cos(th1);
	double S=2*_kmax-sin(th0)+sin(th1);

	double temp1=0.125*(6-4*pow2(_kmax)+2*cos(th0-th1)+4*_kmax*(sin(th0)-sin(th1)));

	if (fabs(temp1)-CUDA_Epsi>1.0){
	  ret[0]=-1;
	  return;
	}

	double invK=1/_kmax;
	double sc_s2 = mod2pi(2*M_PI-acos(temp1))*invK;
	double sc_s1 = mod2pi(th0-atan2(C, S)+0.5*_kmax*sc_s2)*invK;
	double sc_s3 = mod2pi(th0-th1+_kmax*(sc_s2-sc_s1))*invK;

	ret[0]=sc_s1;
	ret[1]=sc_s2;
	ret[2]=sc_s3;
	// printf("in_RLR_dev th0: %f, th1: %f kmax: %f C: %f S: %f temp1: %f invk: %f ret: %f %f %f\n", th0, th1, _kmax, C, S, temp1, invK, ret[0], ret[1], ret[2]);
}

__global__ void LRL (double th0, double th1, double _kmax, double* ret)
{
	double C=cos(th1)-cos(th0);
	double S=2*_kmax+sin(th0)-sin(th1);

	double temp1=0.125*(6-4*pow2(_kmax)+2*cos(th0-th1)-4*_kmax*(sin(th0)-sin(th1)));

	if (fabs(temp1)-CUDA_Epsi>1.0){
	  ret[0]=-1;
	  return;
	}

	double invK=1/_kmax;
	double sc_s2 = mod2pi(2*M_PI-acos(temp1))*invK;
	double sc_s1 = mod2pi(atan2(C, S)-th0+0.5*_kmax*sc_s2)*invK;
	double sc_s3 = mod2pi(th1-th0+_kmax*(sc_s2-sc_s1))*invK;

	ret[0]=sc_s1;
	ret[1]=sc_s2;
	ret[2]=sc_s3;
	// printf("in_LRL_dev th0: %f, th1: %f kmax: %f C: %f S: %f temp1: %f invk: %f ret: %f %f %f\n", th0, th1, _kmax, C, S, temp1, invK, ret[0], ret[1], ret[2]);
}

static double sincCuda(double t) {
  if (std::abs(t)<0.002)
    return 1 - pow2(t)/6 * (1 - pow2(t)/20);
  else
    return sin(t)/t;
}

Configuration2<double> circlineCuda(double _L,
                                Configuration2<double> _P0,
                                double _K)
{
  double app=_K*_L/2.0;
  double sincc=_L*sincCuda(app);
  double phi=_P0.angle().toRad();
  
  double x=_P0.x() + sincc * cos(phi+app);
  double y=_P0.y() + sincc * sin(phi+app);
  Angle th=Angle(_K*_L+phi, Angle::RAD);	

  return Configuration2<double>(x, y, th);
}

class DubinsArcCuda {
public:
	double x0, y0, th0;
	double x1, y1, th1;
	double L, K;

	DubinsArcCuda(double _x0, double _y0, double _th0,
						double _x1, double _y1, double _th1, 
						double _L, double _K) : x0(_x0), y0(_y0), th0(_th0), 
						x1(_x1), y1(_y1), th1(_th1), L(_L), K(_K) {}

	DubinsArcCuda(Configuration2<double> start, double kmax, double _L) : L(_L), K(kmax) {
		x0=start.point().x();
		y0=start.point().y();
		th0=start.angle().toRad();
		Configuration2<double> end=circlineCuda(L, start, kmax);
		x1=end.point().x();
		y1=end.point().y();
		th1=end.angle().toRad();
	}

	void print() {
		printf("x0: %f, y0: %f, th0: %f, x1: %f, y1: %f, th1: %f, L: %f, K: %f\n", 
						x0, y0, th0, x1, y1, th1, L, K);
	}

	Configuration2<double> start(){
		return Configuration2<double> (x0, y0, Angle(th0, Angle::RAD));
	}

	Configuration2<double> end(){
		return Configuration2<double> (x1, y1, Angle(th1, Angle::RAD));
	}
};

void shortest_cuda(	double x0, double y0, double th0, 
										double x1, double y1, double th1, 
										double _kmax=1){
	double Length=DInf;
	int pidx=-1;
	double sc_s[3];
	
	//Scale to standard
	double phi=mod2pi(atan2((y1-y0), (x1-x0)));
	double lambda=sqrt(pow2((y1-y0))+pow2((x1-x0)))/2.0;
	double sc_th0=mod2pi(th0-phi);
	double sc_th1=mod2pi(th1-phi);
	double sc_Kmax=_kmax*lambda;

#ifdef STREAMS
	double** ret=(double**) malloc(sizeof(double*)*6);
	for(int i=0; i<6; i++){
		ret[i]=(double*) malloc(sizeof(double)*3);
	}
	double* dev_RSR; hipMalloc((void**)&dev_RSR, 3*sizeof(double));
	double* dev_LSR; hipMalloc((void**)&dev_LSR, 3*sizeof(double));
	double* dev_RSL; hipMalloc((void**)&dev_RSL, 3*sizeof(double));
	double* dev_RLR; hipMalloc((void**)&dev_RLR, 3*sizeof(double));
	double* dev_LRL; hipMalloc((void**)&dev_LRL, 3*sizeof(double));
	double* dev_LSL; hipMalloc((void**)&dev_LSL, 3*sizeof(double));

	hipStream_t stream[6];
	hipStreamCreate(&stream[0]);
	hipStreamCreate(&stream[1]);
	hipStreamCreate(&stream[2]);
	hipStreamCreate(&stream[3]);
	hipStreamCreate(&stream[4]);
	hipStreamCreate(&stream[5]);

	RSR<<<1, 1, 0, stream[0]>>>(sc_th0, sc_th1, sc_Kmax, dev_RSR);
	hipMemcpyAsync(ret[0], dev_RSR, sizeof(double)*3, hipMemcpyDeviceToHost, stream[0]);
	LSR<<<1, 1, 0, stream[1]>>>(sc_th0, sc_th1, sc_Kmax, dev_LSR);
	hipMemcpyAsync(ret[1], dev_LSR, sizeof(double)*3, hipMemcpyDeviceToHost, stream[1]);
	RSL<<<1, 1, 0, stream[2]>>>(sc_th0, sc_th1, sc_Kmax, dev_RSL);
	hipMemcpyAsync(ret[2], dev_RSL, sizeof(double)*3, hipMemcpyDeviceToHost, stream[2]);
	RLR<<<1, 1, 0, stream[3]>>>(sc_th0, sc_th1, sc_Kmax, dev_RLR);
	hipMemcpyAsync(ret[3], dev_RLR, sizeof(double)*3, hipMemcpyDeviceToHost, stream[3]);
	LRL<<<1, 1, 0, stream[4]>>>(sc_th0, sc_th1, sc_Kmax, dev_LRL);
	hipMemcpyAsync(ret[4], dev_LRL, sizeof(double)*3, hipMemcpyDeviceToHost, stream[4]);
	LSL<<<1, 1, 0, stream[5]>>>(sc_th0, sc_th1, sc_Kmax, dev_LSL);
	hipMemcpyAsync(ret[5], dev_LSL, sizeof(double)*3, hipMemcpyDeviceToHost, stream[5]);

	hipStreamDestroy(stream[0]);
	hipStreamDestroy(stream[1]);
	hipStreamDestroy(stream[2]);
	hipStreamDestroy(stream[3]);
	hipStreamDestroy(stream[4]);
	hipStreamDestroy(stream[5]);

  hipFree(dev_RSR);
	hipFree(dev_LSR);
	hipFree(dev_RSL);
	hipFree(dev_RLR);
	hipFree(dev_LRL);
	hipFree(dev_LSL);

	for(int i=0; i<6; i++){
		double* value=ret[i];
		if (value[0]!=-1){
		  double appL=value[0]+value[1]+value[2];
		  if (appL<Length && !equal(appL, 0.0)){
		    Length = appL;
		    sc_s[0]=value[0];
		    sc_s[1]=value[1];
		    sc_s[2]=value[2];
		    pidx=i;
		  }
		}
  }

  for (int i=0; i<6; i++){
  	free(ret[i]);
  }
  free(ret);
#else
	double* ret=(double*) malloc(sizeof(double)*18);

	size_t pitch;
	double* dev_ret; hipMallocPitch(&dev_ret, &pitch, 3*sizeof(double), 6);

	LSL<<<1, 1>>>(sc_th0, sc_th1, sc_Kmax, dev_ret);
	RSR<<<1, 1>>>(sc_th0, sc_th1, sc_Kmax, dev_ret+1*pitch/sizeof(double));
	LSR<<<1, 1>>>(sc_th0, sc_th1, sc_Kmax, dev_ret+2*pitch/sizeof(double));
	RSL<<<1, 1>>>(sc_th0, sc_th1, sc_Kmax, dev_ret+3*pitch/sizeof(double));
	RLR<<<1, 1>>>(sc_th0, sc_th1, sc_Kmax, dev_ret+4*pitch/sizeof(double));
	LRL<<<1, 1>>>(sc_th0, sc_th1, sc_Kmax, dev_ret+5*pitch/sizeof(double));

	hipMemcpy2D(ret, 3*sizeof(double), dev_ret, pitch, 3*sizeof(double), 6, hipMemcpyDeviceToHost);

	hipFree(dev_ret);

	int a=0;
	// printf("RSR %f %f %f\n", ret[a+0], ret[a+1], ret[a+2]); a+=3;
	// printf("LSR %f %f %f\n", ret[a+0], ret[a+1], ret[a+2]); a+=3;
	// printf("RSL %f %f %f\n", ret[a+0], ret[a+1], ret[a+2]); a+=3;
	// printf("RLR %f %f %f\n", ret[a+0], ret[a+1], ret[a+2]); a+=3;
	// printf("LRL %f %f %f\n", ret[a+0], ret[a+1], ret[a+2]); a+=3;
	// printf("LSL %f %f %f\n", ret[a+0], ret[a+1], ret[a+2]); 

	for(int i=0; i<6; i++){
		double* value=ret+i*3;
		if (value[0]!=-1){
		  double appL=value[0]+value[1]+value[2];
		  if (appL<Length && !equal(appL, 0.0)){
		    Length = appL;
		    sc_s[0]=value[0];
		    sc_s[1]=value[1];
		    sc_s[2]=value[2];
		    pidx=i;
		  }
		}
  }
  if (pidx>=0){
  	//Scale back
  	double sc_std0=sc_s[0]*lambda;
  	double sc_std1=sc_s[1]*lambda;
  	double sc_std2=sc_s[2]*lambda;
  	double ksings[6][3]={
      { 1,  0,  1}, // LSL
      {-1,  0, -1}, // RSR
      { 1,  0, -1}, // LSR
      {-1,  0,  1}, // RSL
      {-1,  1, -1}, // RLR
      { 1, -1,  1}  // LRL
    };

    DubinsArcCuda A0 (Configuration2<double>(x0, y0, Angle(th0, Angle::RAD)), ksings[pidx][0]*_kmax, sc_std0);
    DubinsArcCuda A1 (A0.end(), ksings[pidx][1]*_kmax, sc_std1);
    DubinsArcCuda A2 (A1.end(), ksings[pidx][2]*_kmax, sc_std2);
    
    COUT(pidx)
    A0.print();
    A1.print();
    A2.print();
    cout << endl;
  }
  
  free(ret);

#endif
}


#define GRID 1
#define THREADS 256
//TODO test implementation where x_i=y%base^i
__device__ __host__ 
void toBase(double* v, const double* angles, const double* inc, 
						const int base, int value, size_t size, int startPos, int endPos){
	for (int i=0; i<size; i++){
		if (i<startPos || i>endPos){
			v[i]=angles[i];
		}
		else{
			v[i]=(value%base)*(*inc)+angles[i];
			value=(int)(value/base);
		}
	}
}

// __device__ void dubins(	double* x0, double* y0, double th0,
// 													double* x1, double* y1, double th1, 
// 													double _kmax, double* L, int* id){
// 	int pidx=-1;
// 	//Scale to standard
// 	printf("x0: %f\n", *x0);
// 	printf("atan2: %f\n", atan2(((*y1)-(*y0)), ((*x1)-(*x0))));
// 	double sc_th0=mod2pi((th0)-atan2(((*y1)-(*y0)), ((*x1)-(*x0))));
// 	printf("ciao\n");
// 	double sc_th1=mod2pi((th1)-atan2(((*y1)-(*y0)), ((*x1)-(*x0))));
// 	double sc_lambda=sqrt(pow2((*y1)-(*y0))+pow2((*x1)-(*x0)))/2;
// 	double sc_kmax=_kmax*sc_lambda;

// 	double Length=CUDA_DInf;
// 	// double sc_s1=0.0;
// 	// double sc_s2=0.0;
// 	// double sc_s3=0.0;

// 	double* ret=new double [18];

// 	LSL<<<1, 1>>>(sc_th0, sc_th1, sc_kmax, ret);
// 	RSR<<<1, 1>>>(sc_th0, sc_th1, sc_kmax, ret+3*sizeof(double));
// 	LSR<<<1, 1>>>(sc_th0, sc_th1, sc_kmax, ret+6*sizeof(double));
// 	RSL<<<1, 1>>>(sc_th0, sc_th1, sc_kmax, ret+9*sizeof(double));
// 	RLR<<<1, 1>>>(sc_th0, sc_th1, sc_kmax, ret+12*sizeof(double));
// 	LRL<<<1, 1>>>(sc_th0, sc_th1, sc_kmax, ret+15*sizeof(double));
// 	printf("%f\n", ret[0]);
// 	printf("%f\n", ret[1]);
// 	printf("%f\n", ret[2]);

// 	__syncthreads();

// 	for (int i=0; i<6; i++){
// 		double* value=ret+i*3*sizeof(double);
// 		if (value[0]!=-1){
// 			double appL=value[0]+value[1]+value[2];
// 			if (appL<Length){
// 				Length=appL;
// 				// sc_s1=value[0];
// 				// sc_s1=value[1];
// 				// sc_s1=value[2];
// 				pidx=i;
// 			}
// 		}
// 	}
// 	printf("ciao:\n");
// 	printf("%f\n", Length);
// 	*L=Length;
// 	*id=pidx;

	// if (pidx>=0){
	// 	//Scale from standard
	// 	sc_s1*=sc_lambda;
	// 	sc_s2*=sc_lambda;
	// 	sc_s3*=sc_lambda;

	// 	int ksings[6][3]={
	//      { 1,  0,  1}, // LSL
	//      {-1,  0, -1}, // RSR
	//      { 1,  0, -1}, // LSR
	//      {-1,  0,  1}, // RSL
	//      {-1,  1, -1}, // RLR
	//      { 1, -1,  1}  // LRL
	//    };

	//    #define L sc_s1
	//    double K=_kmax*ksings[pidx][0];
	//    double app=K*L/2.0;
	//    double sincc=L*sinc(app);
	//    dubinsArc A1(	x0, y0, th0, L, 
	//    						(x0+sincc*cos(app)), (y0+sincc+sin(app)),
	//    						mod2pi(K*L+th0), K);
	  
	  
	//    #define L sc_s2
	//    K=_kmax*ksings[pidx][1];
	//    app=K*L/2.0;
	//    sincc=L*sinc(app);
	//    dubinsArc A1(	A0.x1, A0.y1, A0.th1, 
	//    							(A0.x1+sincc*cos(app)), (A0.y1+sincc+sin(app)),
	//    							mod2pi(_kmax*L+A0.th1), 
	//    							L, ksings[pidx][0]*_kmax);

	//   	#define L sc_s3
	//    K=_kmax*ksings[pidx][3];
	//    app=K*L/2.0;
	//    sincc=L*sinc(app);
	//    dubinsArc A2(	A1.x1, A1.y1, A1.th1, 
	//    							(A1.x1+sincc*cos(app)), (A1.y1+sincc+sin(app)),
	//    							mod2pi(_kmax*L+A1.th1), 
	//    							L, ksings[pidx][0]*_kmax); 
	// }
// }

#if __CUDA_ARCH__ < 600
__device__ double myAtomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);
    
    return __longlong_as_double(old);
}
#endif

__global__ void dubins (const double* x, const double* y, double* th, double* length, double* kmax, uint old){
	double _kmax=*kmax;
	uint tidx=threadIdx.x;

	double x0=x[tidx];
	double x1=x[tidx+1];
	double y0=y[tidx];
	double y1=y[tidx+1];
	double th0=th[tidx];
	double th1=th[tidx+1];

	// printf("[%u] x0: %f y0: %f th0: %f x1: %f y1: %f th1: %f\n", tidx, x0, y0, th0, x1, y1, th1);

	int pidx=-1;
	//Scale to standard
	double sc_th0=mod2pi(th0-atan2((y1-y0), (x1-x0)));
	double sc_th1=mod2pi(th1-atan2((y1-y0), (x1-x0)));
	double sc_lambda=sqrt(pow2((y1-y0))+pow2((x1-x0)))/2.0;
	double sc_kmax=_kmax*sc_lambda;

	double Length=CUDA_DInf;
	printf("[%u] pow2(y1-y0)=%f pow2(x1-x0)=%f sum=%f sqrt()=%f\nsc_th0: %f sc_th1: %f sc_lambda: %f sc_kmax: %f Length: %f\n", tidx,
				(pow2((y1-y0))), (pow2((x1-x0))), (pow2((y1-y0))+pow2((x1-x0))), sqrt(pow2((y1-y0))+pow2((x1-x0))),
 				sc_th0, sc_th1, sc_lambda, sc_kmax, Length);
	double sc_s1=0.0;
	double sc_s2=0.0;
	double sc_s3=0.0;

	double* ret=(double*) malloc(sizeof(double)*18);

	LSL<<<1, 1>>>(sc_th0, sc_th1, sc_kmax, &(ret[0]));
	RSR<<<1, 1>>>(sc_th0, sc_th1, sc_kmax, &(ret[3]));
	LSR<<<1, 1>>>(sc_th0, sc_th1, sc_kmax, &(ret[6]));
	RSL<<<1, 1>>>(sc_th0, sc_th1, sc_kmax, &(ret[9]));
	RLR<<<1, 1>>>(sc_th0, sc_th1, sc_kmax, &(ret[12]));
	LRL<<<1, 1>>>(sc_th0, sc_th1, sc_kmax, &(ret[15]));
	hipError_t err=hipDeviceSynchronize();
	if (err!=hipSuccess){
		printf("Error: %d\n", err);
	}

	for (int i=0; i<6; i++){
		double* value=&ret[i*3];
		if (!CUDA_equal(value[0], -1.0)){
			double appL=(value[0]+value[1]+value[2])*sc_lambda;
			if (old==0){
				// printf("[%u] [%u] %d values %f %f %f\n", old, tidx, i, value[0], value[1], value[2]);
			}
			if (appL<Length){
				Length=appL;
				pidx=i;
			}
		}
		else {
			// printf("[%u] [%u] %d Nope %f %f %f\n", old, tidx, i, value[0], value[1], value[2]);
		}
	}
	if (Length<CUDA_DInf){
	  printf("[%u] [%u] Length: %f {%f %f %f}\n", old, tidx, Length, ret[pidx*3], ret[pidx*3+1], ret[pidx*3+2]);
	  // printf("[%u] [%u] x0: %f y0: %f th0: %f x1: %f y1: %f th1: %f Length %f, length %f length %p\n", old, tidx, x0, y0, th0, x1, y1, th1, Length, length[0], length);
	  hipDeviceSynchronize();
	  
	  #if __CUDA_ARCH__ < 600
		  myAtomicAdd(length, Length);
		#else 
		  atomicAdd(length, Length);
	  #endif
	  // printf("[%u] [%u] x0: %f y0: %f th0: %f x1: %f y1: %f th1: %f Length %f, length %f\n", old, tidx, x0, y0, th0, x1, y1, th1, Length, length[0]);
	}
	free(ret);
}

// __global__ void prova(double* x, double* y, double* angles, double* length, double* kmax){
// 	uint tidx=blockDim.x*blockIdx.x+threadIdx.x;
// 	angles[tidx]=100.0;
// 	angles[tidx+1]=100.0;
// 	x[tidx]=100.0;
// 	x[tidx+1]=100.0;
// 	y[tidx]=100.0;
// 	y[tidx+1]=100.0;
// 	MyatomicAdd(length, *kmax);
// }


__global__ void computeDubins (double* _angle, double* inc, double* x, double* y,
															double* lengths, uint* dev_iter, size_t size, size_t base, double* _kmax){
	uint tidx=blockDim.x*blockIdx.x+threadIdx.x;
	if (tidx>=(*dev_iter)){}
	else {
		printf("ciao\n");
		double* angles=(double*) malloc(sizeof(double)*size);
		toBase(angles, _angle, inc, base, tidx, size, 1, size-2);

		// printf("[%d] inc: %f size %u\n", tidx, (*inc), size);
		// printf("[%d] angle+inc: %f, %f, %f, %f, %f\n", tidx, (angles[0]*(*inc)), (angles[1]*(*inc)), (angles[2]*(*inc)), (angles[3]*(*inc)), (angles[4]*(*inc)));
		// printf("[%d] angle: %f, %f, %f, %f, %f\n", tidx, (angles[0]), (angles[1]), (angles[2]), (angles[3]), (angles[4]));
		// printf("[%d] init: %f, %f, %f, %f, %f\n", tidx, _angle[0], _angle[1], _angle[2], _angle[3], _angle[4]);
		// printf("[%d] all: %f, %f, %f, %f, %f\n", tidx, (angles[0]*(*inc)+_angle[0]), (angles[1]*(*inc)+_angle[1]), (angles[2]*(*inc)+_angle[2]), (angles[3]*(*inc)+_angle[3]), (angles[4]*(*inc)+_angle[4]));
		// printf("[%d] _angle: %p inc: %p x: %p y: %p lengths: %p angles: %p \n", tidx, &(_angle[0]), inc, x, y, lengths, angles);
		
		// printf("[%u] angles[0]: %p, angles[1]: %p, angles[2]: %p, angles[3]: %p, angles[4]: %p\n", tidx, &angles[0], &angles[1], &angles[2], &angles[3], &angles[4]);
		// printf("[%u] angles[0]: %f, angles[1]: %f, angles[2]: %f, angles[3]: %f, angles[4]: %f\n", tidx, angles[0], angles[1], angles[2], angles[3], angles[4]);

		// prova<<<1, size-1>>> (x, y, angles, lengths+tidx, _skmax);
		dubins<<<1, size-1>>>(x, y, angles, lengths+tidx, _kmax, tidx);
		hipDeviceSynchronize();
		
		// printf("[%u] angles[0]: %p, angles[1]: %p, angles[2]: %p, angles[3]: %p, angles[4]: %p\n", tidx, &angles[0], &angles[1], &angles[2], &angles[3], &angles[4]);
		printf("[%u] angles[0]: %f, angles[1]: %f, angles[2]: %f, angles[3]: %f\n", tidx, angles[0], angles[1], angles[2], angles[3]);
		// printf("[%u] x[0]: %p, x[1]: %p, x[2]: %p, x[3]: %p, x[4]: %p\n", tidx, &x[0], &x[1], &x[2], &x[3], &x[4]);
		// printf("[%u] x[0]: %f, x[1]: %f, x[2]: %f, x[3]: %f, x[4]: %f\n", tidx, x[0], x[1], x[2], x[3], x[4]);
		// printf("[%u] y[0]: %p, y[1]: %p, y[2]: %p, y[3]: %p, y[4]: %p\n", tidx, &y[0], &y[1], &y[2], &y[3], &y[4]);
		// printf("[%u] y[0]: %f, y[1]: %f, y[2]: %f, y[3]: %f, y[4]: %f\n", tidx, y[0], y[1], y[2], y[3], y[4]);
		printf("[%u] length: %f %p\n", tidx, lengths[tidx], &lengths[tidx]);
		// for (int i=0; i<size-1; i++){
		// 	dubins(x+i*sizeof(double), y+i*sizeof(double), angles[i]*(*inc)+_angle[i], 
		// 				x+(i+1)*sizeof(double), y+(i+1)*sizeof(double), angles[i+1]*(*inc)+_angle[i+1], 
		// 				_kmax, &lengths[tidx], &pidxs[tidx]);
		// 	printf("[%d] [%d] length: %f\n", tidx, i, lengths[tidx]);
		// }
		// printf("prova %f\n", lengths[0]);
		free(angles);
	}
}

double* dubinsSetBest(Configuration2<double> start,
										Configuration2<double> end,
										Tuple<Point2<double> > _points,
										int startPos,
										int endPos,
										uint parts, 
										double _kmax){
	size_t size=_points.size()+2;
	COUT(parts)
	COUT(size)
	unsigned long M=size-startPos;
  if (endPos>startPos){
    M-=(size-endPos-1);
  }
  COUT(M)
  COUT(parts)
  unsigned long iter_n=pow(parts, M);
	COUT(iter_n)

	double* init_angle=(double*) malloc(sizeof(double)*size);
	double* x=(double*) malloc(size*sizeof(double));
	double* y=(double*) malloc(size*sizeof(double));

	init_angle[0]=start.angle().toRad();
	x[0]=start.point().x();
	y[0]=start.point().y();
	for (int i=1; i<size-2; i++){
		init_angle[i]=_points.get(i-1).th(_points.get(i)).toRad();
		x[i]=_points.get(i-1).x();
		y[i]=_points.get(i-1).y();
	}
	init_angle[size-2]=_points.get(_points.size()-1).th(end.point()).toRad();
	x[size-2]=_points.get(_points.size()-1).x();
	y[size-2]=_points.get(_points.size()-1).y();
	init_angle[size-1]=end.angle();
	x[size-1]=end.point().x();
	y[size-1]=end.point().y();
	
	for (int i=0; i<size; i++){
		cout << init_angle[i] << (i!=size-1 ? ", " : "\n");
	}
	// for (int i=0; i<size; i++){
	// 	cout << x[i] << (i!=size-1 ? ", " : "\n");
	// }
	// for (int i=0; i<size; i++){
	// 	cout << y[i] << (i!=size-1 ? ", " : "\n");
	// }


	double Length=DInf;
	double* lengths=(double*) malloc(sizeof(double)*iter_n);
	int* pidxs=(int*) malloc(sizeof(int)*iter_n);
	double inc=2.0*M_PI/parts;

	double* dev_x; hipMalloc((void**)&dev_x, sizeof(double)*size);
	double* dev_y; hipMalloc((void**)&dev_y, sizeof(double)*size);
	double* dev_init_angle; hipMalloc((void**)&dev_init_angle, sizeof(double)*size);

	double* dev_lengths; hipMalloc((void**)&dev_lengths, sizeof(double)*iter_n); 
	uint* dev_iter; hipMalloc((void**)&dev_iter, sizeof(uint));
	double* dev_inc; hipMalloc((void**)&dev_inc, sizeof(double)); 
	double* dev_kmax; hipMalloc((void**)&dev_kmax, sizeof(double)); 
	// int* dev_pidxs; hipMalloc((void**)&dev_pidxs, sizeof(int)*iter_n);

	hipMemcpy(dev_x, x, sizeof(double)*size, hipMemcpyHostToDevice);
	hipMemcpy(dev_y, y, sizeof(double)*size, hipMemcpyHostToDevice);
	hipMemcpy(dev_init_angle, init_angle, sizeof(double)*size, hipMemcpyHostToDevice);
	
	hipMemcpy(dev_iter, &iter_n, sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy(dev_inc, &inc, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_kmax, &_kmax, sizeof(double), hipMemcpyHostToDevice);

	computeDubins<<<((int)(iter_n/THREADS)+1), (THREADS>iter_n ? iter_n : THREADS)>>> 
	// computeDubins<<<1, 2>>> 
												(dev_init_angle, dev_inc, dev_x, dev_y, 
												dev_lengths, dev_iter, size, parts, dev_kmax);
	int val=hipDeviceSynchronize();
	if (val!=hipSuccess)
		printf("After dubins: %d\n", val);
	else 
		printf("val: %d %d\n", val, (int)hipSuccess);

	hipMemcpy(lengths, dev_lengths, sizeof(double)*iter_n, hipMemcpyDeviceToHost);
	// hipMemcpy(pidxs, dev_pidxs, sizeof(double)*iter_n, hipMemcpyDeviceToHost);

	int pidx=-1;
	for (int i=0; i<iter_n; i++){
		if (lengths[i]<Length && !equal(lengths[i], 0)){
			Length=lengths[i];
			pidx=i;
		}
	}

	COUT(Length)
	COUT(pidx)

	double* angls=(double*) malloc(sizeof(double)*size);
	toBase(angls, init_angle, &inc, parts, pidx, size, 1, size-2);
	for (int i=0; i<size; i++){
		cout << angls[i] << (i!=size-1 ? ", " : "\n");
	}

	for (int i=0; i<size-1; i++){
		shortest_cuda(x[i], y[i], angls[i], x[i+1], y[i+1], angls[i+1]);
	}

	free(pidxs);
	free(lengths);
	// free(angls);

	hipFree(dev_lengths);
	hipFree(dev_iter);
	hipFree(dev_inc);
	return angls;
}
